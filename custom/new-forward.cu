#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define cudaErrChk(stmt) \
  { cudaAssert((stmt), __FILE__, __LINE__); }

inline void cudaAssert(hipError_t error,
                       const char* file,
                       int line,
                       bool abort = true) {
  if (error != hipSuccess) {
    std::cerr << "CUDA error: "
              << hipGetErrorString(error) << ' ' << file << ':' << line << std::endl;
    if (abort) {
      exit(error);
    }
  }
}

// Some feature flags
//#define USE_STREAM // Use multi-stream to accelerate transfers
//#define USE_ASYNC_ALLOCATOR // Use async allocators, available >= 11.2

// Allocate maximal possible kernel size and reuse it between op1/2
#define M_MAX 16
#define C_MAX 4
#define KERNEL_WIDTH 7
__constant__ float kernel[M_MAX * C_MAX * KERNEL_WIDTH * KERNEL_WIDTH];

// Tile configurations
#define TILE_WIDTH 8
#define PADDED_TILE_WIDTH (TILE_WIDTH + KERNEL_WIDTH - 1)

// Block size along the B (batch) dimension
#define B_BATCH 4

// Prepare input features as column matrix
__global__ void im2col(float* xc,
                       const float* x,
                       const int B,
                       const int M,
                       const int C,
                       const int H,
                       const int W,
                       const int K) {
  extern __shared__ float tile[];

  /*
  Function paramter definitions:
  y - output
  x - input
  k - kernel
  B - batch_size (number of images in x)
  M - number of output feature maps
  C - number of input feature maps
  H - input height dimension
  W - input width dimension
  K - kernel height and width (K x K)
  */

  const int H_out = H - K + 1;
  const int W_out = W - K + 1;

  // Alias for block/thread index
  const int tx = threadIdx.x, ty = threadIdx.y;
  // Alias for batch axis
  const int tb = threadIdx.z;
  const int b = blockIdx.z * blockDim.z + tb;

  // Y = (H W) * (K^2 C)
  /*
      c - input feature map
  ho/wo - output height/width
  hi/wi - input height/width
  hk/wk - convolution loop height/width
  */
#define xc5d(i_ho, i_wo, i_c, i_hk, i_wk)        \
  xc[(b) * (H_out * W_out * C * K * K) +         \
     ((i_ho) * (W_out) + (i_wo)) * (C * K * K) + \
     (i_c) * (K * K) + (i_hk) * (K) + (i_wk)]
#define t3d(i_c, i_ph, i_pw)                                \
  tile[(tb) * (C * PADDED_TILE_WIDTH * PADDED_TILE_WIDTH) + \
       (i_c) * (PADDED_TILE_WIDTH * PADDED_TILE_WIDTH) +    \
       (i_ph) * (PADDED_TILE_WIDTH) +                       \
       (i_pw)]
#define x3d(i_c, i_hi, i_wi) \
  x[(b) * (C * H * W) +      \
    (i_c) * (H * W) +        \
    (i_hi) * (W) +           \
    (i_wi)]

  const int n_bx = (W_out + (TILE_WIDTH - 1)) / TILE_WIDTH;
  const int bx = blockIdx.x % n_bx;
  const int by = blockIdx.x / n_bx;

  if (b < B) {
    if (blockIdx.y == 0) {
      int dst_x, dst_y, src_x, src_y;

      // Pre-load to shared memory, need to loop multiple time, PW^2 / W^2
      for (int c = 0; c < C; c++) {
        for (int dst = ty * TILE_WIDTH + tx;
             dst < PADDED_TILE_WIDTH * PADDED_TILE_WIDTH;
             dst += TILE_WIDTH * TILE_WIDTH) {
          // 3D index inside a padded tiles
          dst_x = dst % PADDED_TILE_WIDTH;
          dst_y = dst / PADDED_TILE_WIDTH;
          // 3D index in global array, simply subtract the pad size
          src_x = (bx * TILE_WIDTH + dst_x);
          src_y = (by * TILE_WIDTH + dst_y);

          if ((src_x < W) && (src_y < H)) {
            t3d(c, dst_y, dst_x) = x3d(c, src_y, src_x);
          } else {
            t3d(c, dst_y, dst_x) = 0.0f;
          }
        }
      }
      __syncthreads();

      // Update destination location
      dst_x = bx * TILE_WIDTH + tx;
      dst_y = by * TILE_WIDTH + ty;

      // Flatten out the matrix for current output pixel
      if ((dst_x < W_out) && (dst_y < H_out)) {
        for (int c = 0; c < C; c++) {
          for (int p = 0; p < KERNEL_WIDTH; p++) {
            for (int q = 0; q < KERNEL_WIDTH; q++) {
              xc5d(dst_y, dst_x, c, p, q) = t3d(c, ty + p, tx + q);
            }
          }
        }
      }
    }
  }

#undef xc5d
#undef t3d
#undef x3d
}

__global__ void matrix_multiply(float* y,
                                const float* xc,
                                const int B,
                                const int M,
                                const int C,
                                const int H,
                                const int W,
                                const int K) {
  extern __shared__ float tile[];

  const int H_out = H - K + 1;
  const int W_out = W - K + 1;

  // Alias for block/thread index
  const int tx = threadIdx.x, ty = threadIdx.y;
  // Alias for batch axis
  const int tb = threadIdx.z;
  const int b = blockIdx.z * blockDim.z + tb;

  /*
   y.shape = (B, M, H_out, W_out)
  xc.shape = (H_out * W_out, C * K * K) = (H_out, W_out, C * K * K)
   t.shape = (b, 0/1, H_out, W_out, C, K, K)
   k.shape = (M, C * K * K)
  */
#define y2d(i_m, i_hw)          \
  y[(b) * (M * H_out * W_out) + \
    (i_m) * (H_out * W_out) +   \
    (i_hw)]
#define xc2d(i_hw, i_ckk)                \
  xc[(b) * (H_out * W_out * C * K * K) + \
     (i_hw) * (C * K * K) +              \
     (i_ckk)]
#define t2d(i, i_hw, i_ckk)                   \
  tile[(tb) * (2 * TILE_WIDTH * TILE_WIDTH) + \
       (i) * (TILE_WIDTH * TILE_WIDTH) +      \
       (i_hw) * (TILE_WIDTH) +                \
       (i_ckk)]
#define k2d(i_m, i_ckk)        \
  kernel[(i_m) * (C * K * K) + \
         (i_ckk)]

#define t2d_xc(i_hw, i_ckk) \
  t2d(0, i_hw, i_ckk)
#define t2d_kt(i_m, i_ckk) \
  t2d(1, i_m, i_ckk)

  const int n_hw = H_out * W_out;

  // Identify the row/column of output element
  // TODO currently, share TILE_WIDTH with im2col, Tensor needs TILE_WIDTH=16
  const int dst_hw = blockIdx.x * TILE_WIDTH + tx;  // col
  const int dst_m = blockIdx.y * TILE_WIDTH + ty;   // row

  // Calculate number of subtiles
  const int n_kernel = C * K * K;
  const int n_tiles = (n_kernel + (TILE_WIDTH - 1)) / TILE_WIDTH;

  if (b < B) {
    int dst_ckk;
    float acc = 0;
    for (int n = 0; n < n_tiles; n++) {
      // Save sub-tile of xc and kernel to smem
      dst_ckk = n * TILE_WIDTH + ty;
      if ((dst_hw < n_hw) && (dst_ckk < n_kernel)) {
        t2d_xc(ty, tx) = xc2d(dst_hw, dst_ckk);
      } else {
        t2d_xc(ty, tx) = 0.0;
      }
      dst_ckk = n * TILE_WIDTH + tx;
      if ((dst_m < M) && (dst_ckk < n_kernel)) {
        t2d_kt(ty, tx) = k2d(dst_m, dst_ckk);
      } else {
        t2d_kt(ty, tx) = 0.0;
      }
      __syncthreads();

      // C_ij = A_ik * B_kj ===> C_ij^T = B_kj^T * A_ik^T
      for (int k = 0; k < TILE_WIDTH; k++) {
        acc += t2d_kt(ty, k) * t2d_xc(k, tx);
      }
      __syncthreads();
    }

    if ((dst_m < M) && (dst_hw < n_hw)) {
      y2d(dst_m, dst_hw) = acc;
    }
  }
#undef t2d_xc
#undef t2d_kt

#undef y2d
#undef xc2d
#undef t2d
#undef k2d
}

__global__ void conv_as_gemm(float* y,
                             float* xc,
                             const float* x,
                             const int B,
                             const int M,
                             const int C,
                             const int H,
                             const int W,
                             const int K) {
  extern __shared__ float tile[];

  const int H_out = H - K + 1;
  const int W_out = W - K + 1;

  // Alias for height/width axis
  const int t_hw = threadIdx.x;
  // Alias for output channels
  const int t_m = threadIdx.y;
  const int m = blockIdx.y * TILE_WIDTH + t_m;
  // Alias for batch axis
  const int t_b = threadIdx.z;
  const int b = blockIdx.z * B_BATCH + t_b;

  // Y = (H W) * (K^2 C)
  /*
      c - input feature map
  ho/wo - output height/width
  hi/wi - input height/width
  hk/wk - convolution loop height/width
  */

  /*
   y.shape = (B, M, H_out, W_out)
  xc.shape = (H_out * W_out, C * K * K) = (H_out, W_out, C * K * K)
   t.shape = (b, 0/1, H_out, W_out, C, K, K)
   k.shape = (M, C * K * K)
  */
#define y1d(i_hw)               \
  y[(b) * (M * H_out * W_out) + \
    (m) * (H_out * W_out) +     \
    (i_hw)]
#define xc2d(i_hw, i_ckk)                  \
  xc[(b) * (H_out * W_out * C * K * K) + \
     (i_hw) * (C * K * K) +                \
     (i_ckk)]
#define xc4d(i_hw, i_c, i_hk, i_wk) \
  xc2d(i_hw, (i_c) * (K * K) + (i_hk) * (K) + (i_wk))
#define x3d(i_c, i_hi, i_wi) \
  x[(b) * (C * H * W) +      \
    (i_c) * (H * W) +        \
    (i_hi) * (W) +           \
    (i_wi)]

#define t2d(i, i_hw, i_ckk)                    \
  tile[(t_b) * (2 * TILE_WIDTH * TILE_WIDTH) + \
       (i) * (TILE_WIDTH * TILE_WIDTH) +       \
       (i_hw) * (TILE_WIDTH) +                 \
       (i_ckk)]
#define k1d(i_ckk)           \
  kernel[(m) * (C * K * K) + \
         (i_ckk)]

#define t2d_xc(i_hw, i_ckk) \
  t2d(0, i_hw, i_ckk)
#define t2d_kt(i_m, i_ckk) \
  t2d(1, i_m, i_ckk)

  const int n_hw = H_out * W_out;

  // Identify the row/column of output element
  // TODO currently, share TILE_WIDTH with im2col, Tensor needs TILE_WIDTH=16
  const int dst_hw = blockIdx.x * TILE_WIDTH + t_hw;

  // Calculate number of subtiles
  const int n_kernel = C * K * K;
  const int n_tiles = (n_kernel + (TILE_WIDTH - 1)) / TILE_WIDTH;

  if ((b < B)) {
    int dst_ckk;

    /*
    // Only first batch of threads need to fetch the data
    if ((t_m == 0) && (dst_hw < n_hw)) {
      // Calculate destination 3D index
      const int dst_x = dst_hw % W_out;
      const int dst_y = dst_hw / W_out;

      // Unroll region of interest
      for (int c = 0; c < C; c++) {
        for (int p = 0; p < K; p++) {
          for (int q = 0; q < K; q++) {
            // Source unrolled index
            dst_ckk = c * (K * K) + p * (K) + q;
            // 3D index in source array simply adds back the pad
            xc4d(dst_hw, c, p, q) = x3d(c, dst_y + p, dst_x + q);
          }
        }
      }
    }
    __syncthreads();
    */

    // Multiplication
    float acc = 0;
    for (int n = 0; n < n_tiles; n++) {
      // Save sub-tile of xc and kernel to smem
      dst_ckk = n * TILE_WIDTH + t_m;
      if ((dst_hw < n_hw) && (dst_ckk < n_kernel)) {
        //t2d_xc(t_m, t_hw) = xc2d(dst_hw, dst_ckk);

        const int dst_x = dst_hw % W_out;
        const int dst_y = dst_hw / W_out;

        int tmp = dst_ckk;
        const int q = tmp % K;
        tmp /= K;
        const int p = tmp % K;
        const int c = tmp / K;

        t2d_xc(t_m, t_hw) = x3d(c, dst_y + p, dst_x + q);
      } else {
        t2d_xc(t_m, t_hw) = 0.0;
      }
      dst_ckk = n * TILE_WIDTH + t_hw;
      if ((m < M) && (dst_ckk < n_kernel)) {
        t2d_kt(t_m, t_hw) = k1d(dst_ckk);
      } else {
        t2d_kt(t_m, t_hw) = 0.0;
      }
      __syncthreads();

      // C_ij = A_ik * B_kj ===> C_ij^T = B_kj^T * A_ik^T
      for (int k = 0; k < TILE_WIDTH; k++) {
        acc += t2d_kt(t_m, k) * t2d_xc(k, t_hw);
      }
      __syncthreads();
    }

    if ((m < M) && (dst_hw < n_hw)) {
      y1d(dst_hw) = acc;
    }
  }

#undef t2d_xc
#undef t2d_kt

#undef y2d
#undef xc2d
#undef x3d
#undef t2d
#undef k2d
}

__host__ void GPUInterface::conv_forward_gpu_prolog(const float* host_y,
                                                    const float* host_x,
                                                    const float* host_k,
                                                    float** device_y_ptr,
                                                    float** device_x_ptr,
                                                    float** device_xc_ptr,
                                                    const int B,
                                                    const int M,
                                                    const int C,
                                                    const int H,
                                                    const int W,
                                                    const int K) {
  std::cout << "*** constant mem + tiled gemm" << std::endl;
  printf("(B=%d, M=%d, C=%d, H=%d, W=%d, K=%d)\n", B, M, C, H, W, K);

  // Estimat output dimension
  const int H_out = H - K + 1;
  const int W_out = W - K + 1;
  printf("(H_out=%d, W_out=%d)\n", H_out, W_out);

  // Calculate needed bytes for original input
  const size_t bytes_y = (B * M * H_out * W_out) * sizeof(float);
  const size_t bytes_x = (B * C * H * W) * sizeof(float);
  const size_t bytes_k = (M * C * K * K) * sizeof(float);

  // Allocate memory on device
  cudaErrChk(hipMalloc(device_y_ptr, bytes_y));
  cudaErrChk(hipMalloc(device_x_ptr, bytes_x));

  // Copy input data to device
  cudaErrChk(hipMemcpy(*device_x_ptr, host_x, bytes_x, hipMemcpyHostToDevice));

  // Copy kernel weights
  cudaErrChk(hipMemcpyToSymbol(HIP_SYMBOL(kernel), host_k, bytes_k));

  // Repurpose unused device_k as buffer for column matrix xc
  const size_t bytes_xc = (B * (H_out * W_out) * (C * K * K)) * sizeof(float);
  std::cout << "*** Xc.size=" << bytes_xc / 1024 << "KiB" << std::endl;
  cudaErrChk(hipMalloc(device_xc_ptr, bytes_xc));
}

__host__ void GPUInterface::conv_forward_gpu(float* device_y,
                                             const float* device_x,
                                             const float* device_xc_const,
                                             const int B,
                                             const int M,
                                             const int C,
                                             const int H,
                                             const int W,
                                             const int K) {
  // Estimat output dimension
  const int H_out = H - K + 1;
  const int W_out = W - K + 1;

  // Remove the constant qualifier
  float* device_xc = (float*)device_xc_const;

  // Calculate launch size
  dim3 block(TILE_WIDTH, TILE_WIDTH, B_BATCH);
  dim3 grid(ceil((float)H_out * W_out / block.x),
            ceil((float)M / block.y),
            ceil((float)B / block.z));
  printf("*** grid=(x=%d, y=%d, z=%d), block=(x=%d, y=%d, z=%d)\n",
         grid.x, grid.y, grid.z, block.x, block.y, block.z);

  // Determine shared memory size
  size_t smem_size_unroll =
      B_BATCH * C * PADDED_TILE_WIDTH * PADDED_TILE_WIDTH * sizeof(float);
  size_t smem_size_gemm =
      B_BATCH * 2 * TILE_WIDTH * TILE_WIDTH * sizeof(float);
  size_t smem_size =
      (smem_size_unroll > smem_size_gemm) ? smem_size_unroll : smem_size_gemm;
  std::cout << "*** smem.size=" << smem_size / 1024 << "KiB" << std::endl;

  if (0) {
    // Unroll to column matrix
    im2col<<<grid, block, smem_size>>>(device_xc, device_x,
                                       B, M, C, H, W, K);
    // GEMM between weights and xc
    matrix_multiply<<<grid, block, smem_size>>>(device_y, device_xc,
                                                B, M, C, H, W, K);
  } else {
    conv_as_gemm<<<grid, block, smem_size>>>(device_y, device_xc, device_x,
                                             B, M, C, H, W, K);
  }
}

__host__ void GPUInterface::conv_forward_gpu_epilog(float* host_y,
                                                    float* device_y,
                                                    float* device_x,
                                                    float* device_xc,
                                                    const int B,
                                                    const int M,
                                                    const int C,
                                                    const int H,
                                                    const int W,
                                                    const int K) {
  const int H_out = H - K + 1;
  const int W_out = W - K + 1;
  const size_t bytes_y = (B * M * H_out * W_out) * sizeof(float);

  // Ensure kernels are executed
  cudaErrChk(hipDeviceSynchronize());

  // Copy output back to host
  cudaErrChk(hipMemcpy(host_y, device_y, bytes_y, hipMemcpyDeviceToHost));

  // Free device memory
  cudaErrChk(hipFree(device_y));
  cudaErrChk(hipFree(device_x));
  cudaErrChk(hipFree(device_xc));
}

__host__ void GPUInterface::get_device_properties() {
  int deviceCount;
  hipGetDeviceCount(&deviceCount);

  for (int dev = 0; dev < deviceCount; dev++) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    std::cout << "Device " << dev << " name: " << deviceProp.name << std::endl;
    std::cout << "Computational capabilities: "
              << deviceProp.major << "." << deviceProp.minor << std::endl;
    std::cout << "Max Global memory size: " << deviceProp.totalGlobalMem
              << std::endl;
    std::cout << "Max Constant memory size: " << deviceProp.totalConstMem
              << std::endl;
    std::cout << "Max Shared memory size per block: " << deviceProp.sharedMemPerBlock
              << std::endl;
    std::cout << "Max threads per block: " << deviceProp.maxThreadsPerBlock
              << std::endl;
    std::cout << "Max block dimensions: "
              << deviceProp.maxThreadsDim[0] << " x, "
              << deviceProp.maxThreadsDim[1] << " y, "
              << deviceProp.maxThreadsDim[2] << " z" << std::endl;
    std::cout << "Max grid dimensions: "
              << deviceProp.maxGridSize[0] << " x, "
              << deviceProp.maxGridSize[1] << " y, "
              << deviceProp.maxGridSize[2] << " z" << std::endl;
    std::cout << "Warp Size: " << deviceProp.warpSize << std::endl;
  }
}
